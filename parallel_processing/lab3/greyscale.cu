#include "hip/hip_runtime.h"
/* ACADEMIC INTEGRITY PLEDGE                                              */
/*                                                                        */
/* - I have not used source code obtained from another student nor        */
/*   any other unauthorized source, either modified or unmodified.        */
/*                                                                        */
/* - All source code and documentation used in my program is either       */
/*   my original work or was derived by me from the source code           */
/*   published in the textbook for this course or presented in            */
/*   class.                                                               */
/*                                                                        */
/* - I have not discussed coding details about this project with          */
/*   anyone other than my instructor. I understand that I may discuss     */
/*   the concepts of this program with other students and that another    */
/*   student may help me debug my program so long as neither of us        */
/*   writes anything during the discussion or modifies any computer       */
/*   file during the discussion.                                          */
/*                                                                        */
/* - I have violated neither the spirit nor letter of these restrictions. */
/*                                                                        */
/*                                                                        */
/*                                                                        */
/* Signed:_____________________________________ Date:_____________        */
/*                                                                        */
/*                                                                        */
/* 3460:677 CUDA Image Processing lab, Version 1.02, Fall 2016.           */

// Application, read and display bitmap file.

#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include "cpu_bitmap.h"
#include "bitmap_help.h"

#define CHANNELS 4 // we have 4 channels corresponding to ARGB
// The input image is encoded as unsigned characters [0, 255]
__global__ void colorConvert(unsigned char *grayImage, unsigned char *rgbImage)
{
   int x = blockIdx.x;
   int y = blockIdx.y;
   if (x < gridDim.x && y < gridDim.y)
   {
      // get 1D coordinate for the grayscale image
      int grayOffset = (y * gridDim.x) + x;
      // get 1D coordinate for the color image
      int rgbOffset = grayOffset * CHANNELS;
      unsigned char r = rgbImage[rgbOffset];     // red value for pixel
      unsigned char g = rgbImage[rgbOffset + 1]; // green value for pixel
      unsigned char b = rgbImage[rgbOffset + 2]; // blue value for pixel

      // convert to grayscale using the formula given
      float intensity = 0.21f * r + 0.71f * g + 0.07f * b;
      grayImage[rgbOffset + 0] = intensity;
      grayImage[rgbOffset + 1] = intensity;
      grayImage[rgbOffset + 2] = intensity;
   }
}

__host__ void imgProc(unsigned char *map, int size, int width, int height)
{
   // Allocate device memory.
   unsigned char *d_rgbImage, *d_grayImage;
   hipMalloc((void **)&d_rgbImage, size * sizeof(unsigned char));
   hipMalloc((void **)&d_grayImage, size * sizeof(unsigned char));

   // Copy host memory (the bitmap pixel data) to device.
   hipMemcpy(d_rgbImage, map, size * sizeof(unsigned char), hipMemcpyHostToDevice);

   // Create a width-by-height grid of 1-by-1 blocks. Each block corresponds to an individual pixel, whose
   // coordinates are given as blockIdx.x + blockIdx.y * gridDim.x.
   dim3 dimGrid(width, height);
   dim3 dimBlock(1, 1);

   // Invoke a CUDA kernel
   colorConvert<<<dimGrid, dimBlock>>>(d_grayImage, d_rgbImage);

   // Copy results from device to host.
   hipMemcpy(map, d_grayImage, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

   // Deallocate device memory.
   hipFree(d_rgbImage);
   hipFree(d_grayImage);
}

int main(void)
{
   char fname[50];
   FILE *infile;
   unsigned short ftype;
   tagBMFH bitHead;
   tagBMIH bitInfoHead;
   tagRGBQ *pRgb;

   printf("Please enter the .bmp file name: ");
   scanf("%s", fname);
   strcat(fname, ".bmp");
   infile = fopen(fname, "rb");

   if (infile != NULL)
   {
      printf("File open successful.\n");
      fread(&ftype, 1, sizeof(unsigned short), infile);
      if (ftype != 0x4d42)
      {
         printf("File not .bmp format.\n");
         return 1;
      }
      fread(&bitHead, 1, sizeof(tagBMFH), infile);
      fread(&bitInfoHead, 1, sizeof(tagBMIH), infile);
   }
   else
   {
      printf("File open fail.\n");
      return 1;
   }

   if (bitInfoHead.biBitCount < 24)
   {
      long nPlateNum = long(pow(2, double(bitInfoHead.biBitCount)));
      pRgb = (tagRGBQ *)malloc(nPlateNum * sizeof(tagRGBQ));
      memset(pRgb, 0, nPlateNum * sizeof(tagRGBQ));
      int num = fread(pRgb, 4, nPlateNum, infile);
   }

   int width = bitInfoHead.biWidth;
   int height = bitInfoHead.biHeight;
   int l_width = 4 * ((width * bitInfoHead.biBitCount + 31) / 32);
   long nData = height * l_width;
   unsigned char *pColorData = (unsigned char *)malloc(nData);
   memset(pColorData, 0, nData);
   fread(pColorData, 1, nData, infile);

   fclose(infile);

   CPUBitmap dataOfBmp(width, height);
   unsigned char *map = dataOfBmp.get_ptr();

   if (bitInfoHead.biBitCount < 24)
   {
      int k, index = 0;
      if (bitInfoHead.biBitCount == 1)
      {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++)
            {
               unsigned char mixIndex = 0;
               k = i * l_width + j / 8;
               mixIndex = pColorData[k];
               if (j % 8 < 7)
                  mixIndex = mixIndex << (7 - (j % 8));
               mixIndex = mixIndex >> 7;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
      }
      else if (bitInfoHead.biBitCount == 2)
      {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++)
            {
               unsigned char mixIndex = 0;
               k = i * l_width + j / 4;
               mixIndex = pColorData[k];
               if (j % 4 < 3)
                  mixIndex = mixIndex << (6 - 2 * (j % 4));
               mixIndex = mixIndex >> 6;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
      }
      else if (bitInfoHead.biBitCount == 4)
      {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++)
            {
               unsigned char mixIndex = 0;
               k = i * l_width + j / 2;
               mixIndex = pColorData[k];
               if (j % 2 == 0)
                  mixIndex = mixIndex << 4;
               mixIndex = mixIndex >> 4;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
      }
      else if (bitInfoHead.biBitCount == 8)
      {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++)
            {
               unsigned char mixIndex = 0;
               k = i * l_width + j;
               mixIndex = pColorData[k];
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
      }
      else if (bitInfoHead.biBitCount == 16)
      {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++)
            {
               unsigned char mixIndex = 0;
               k = i * l_width + j * 2;
               unsigned char shortTemp = pColorData[k + 1] << 8;
               mixIndex = pColorData[k] + shortTemp;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
      }
   }
   else
   {
      int k, index = 0;
      for (int i = 0; i < height; i++)
         for (int j = 0; j < width; j++)
         {
            k = i * l_width + j * 3;
            map[index * 4 + 0] = pColorData[k + 2];
            map[index * 4 + 1] = pColorData[k + 1];
            map[index * 4 + 2] = pColorData[k];
            index++;
         }
   }

   imgProc(map, dataOfBmp.image_size(), width, height);
   dataOfBmp.display_and_exit();
   return 0;
}